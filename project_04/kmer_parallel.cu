#include "hip/hip_runtime.h"
//
// Created by umityigitbsrn on 7.01.2022.
//

#include "util_parallel.h"
#include <time.h>

int num_kmer_in_read(char *read, int k);

int main(int argc, char** argv)
{
    if(argc != 5) {
        printf("Wrong argments usage: ./kmer_parallel [REFERENCE_FILE] [READ_FILE] [k] [OUTPUT_FILE]\n" );
    }

    FILE *fp;
    int k;

    //malloc instead of allocating in stack
    char *reference_str = (char*) malloc(MAX_REF_LENGTH * sizeof(char));
    char *read_str = (char*) malloc(MAX_READ_LENGTH * sizeof(char));

    char *reference_filename, *read_filename, *output_filename;
    int reference_length;

    reference_filename = argv[1];
    read_filename = argv[2];
    k = atoi(argv[3]);
    output_filename = argv[4];

    fp = fopen(reference_filename, "r");
    if (fp == NULL) {
        printf("Could not open file %s!\n",reference_filename);
        return 1;
    }

    if (fgets(reference_str, MAX_REF_LENGTH, fp) == NULL) { //A single line only
        printf("Problem in file format!\n");
        return 1;
    }
    reference_str[strcspn(reference_str, "\n")] = 0; //Remove the trailing \n character

    reference_length = strlen(reference_str);

//    printf("Reference str is = %s\n", reference_str);
    fclose(fp);

    //Read queries
    StringList queries;

    initStringList(&queries, 3);  // initially 3 elements

    int success = read_file(read_filename,&queries);
//    if(success){
//        for(int i = 0; i < queries.used; i++) {
//            printf("read : %s, len: %zu\n", queries.array[i], strlen(queries.array[i]));
//        }
//    }


    ////////////////////////////////////////////////////////////////////////
    ////////////// THIS IS A GOOD PLACE TO DO YOUR COMPUTATIONS ////////////
    ////////////////////////////////////////////////////////////////////////

    clock_t t;
    t = clock();
    // send from host to device
    int len_read = strlen(queries.array[0]);

    // reference str
    char *dev_ref;
    int size = reference_length * sizeof(char);
    hipMalloc(&dev_ref, size);
    hipMemcpy(dev_ref, reference_str, size, hipMemcpyHostToDevice);

    // string list
//    StringList *dev_str_list;
//    size = sizeof(StringList);
//    hipMalloc(&dev_str_list, size);
//    hipMemcpy(dev_str_list, &queries, size, hipMemcpyHostToDevice);

    char *read_tmp = (char *) malloc(sizeof(char) * len_read * queries.used);
    int tmp_index = 0;
    for (int i = 0; i < queries.used; ++i){
        for (int j = 0; j < len_read; ++j){
            read_tmp[tmp_index] = queries.array[i][j];
            tmp_index++;
        }
    }

//    printf("read_tmp: %s\n", read_tmp);

    char *dev_read;
    size = len_read * sizeof(char) * queries.used;
    hipMalloc(&dev_read, size);
    hipMemcpy(dev_read, read_tmp, size, hipMemcpyHostToDevice);

    // output
    size = sizeof(int) * (len_read - k + 1) * queries.used;
//    size = sizeof(int) * (len_read - k + 1);
    int *dev_out;
    hipMalloc(&dev_out, size);

    // kernel function

    // set thread and block numbers
    
    unsigned int num_of_threads = (len_read - k + 1) * queries.used;
    unsigned int num_of_blocks = num_of_threads / 1024;
    unsigned int remainder = num_of_threads % 1024;
    
    if (remainder > 0)
	    num_of_blocks++;

    dim3 dim_grid(num_of_blocks, 1);
    dim3 dim_block(1024, 1);
//    dim3 dim_block((len_read - k + 1), 1);

//    kernel_fnc<<<dim_grid, dim_block>>>(dev_ref, dev_str_list, dev_out, k, reference_length, len_read);
    kernel_fnc<<<dim_grid, dim_block>>>(dev_ref, dev_read, dev_out, k, reference_length, len_read, num_of_threads);
//    hipDeviceSynchronize();

    // device to host
    int *host_out = (int *) malloc(sizeof(int) * size);
    hipMemcpy(host_out, dev_out, size, hipMemcpyDeviceToHost);

//    for (int i = 0; i < queries.used; ++i){
//        prin f("[ ");
//        for (int j = 0; j < (len_read - k + 1); ++j){
//            printf("%d ", host_out[i * (len_read - k + 1) + j]);
//        }
//        printf("]\n");
//    }


//    for (int i = 0; i < queries.used; ++i){
//        printf("source: %s\n", queries.array[i]);
//        for (int j = k; j <= strlen(queries.array[i]); ++j){
//            substring(read_str, queries.array[i], j - k, j);
//            printf("%s", read_str);
//
//            if (host_out[i * (len_read - k + 1) + (j - k)] != -1){
//                printf("( count=1 at index %d )\n", host_out[i * (len_read - k + 1) + (j - k)]);
//            } else {
//                printf("( count=0 at index -1 )\n");
//            }
//        }
//    }

    t = clock() - t;
    double elapsed_time = ((double) t)/CLOCKS_PER_SEC;
    printf("-------elapsed time - parallel: %f-------\n", elapsed_time);

    write_file(output_filename, host_out, queries.used, len_read - k + 1);

    // f ee cuda
    hipFree(dev_ref);
//    hipFree(dev_str_list);
    hipFree(dev_read);
    hipFree(dev_out);

    //Free up
    freeStringList(&queries);

    free(reference_str);
    free(read_str);
    free(read_tmp);
    free(host_out);
}
