#include "hip/hip_runtime.h"
#include "util_parallel.h"

//void init_list(dyn_list *l, size_t initial_size){
//    l->item_arr = (int *) malloc()
//}

void initStringList(StringList *a, size_t initialSize) {
    a->array = (char**) malloc(initialSize * sizeof(char*));
    for (int i = 0; i < initialSize; i++) {
        a->array[i] = (char*) malloc(MAX_READ_LENGTH * sizeof(char));
    }
    a->used = 0;
    a->size = initialSize;
}

void insertStringList(StringList *a, char *element) {
    // a->used is the number of used entries, because a->array[a->used++] updates a->used only *after* the array has been accessed.
    // Therefore a->used can go up to a->size
    if (a->used == a->size) {
        a->size *= 2;
        a->array = (char**) realloc(a->array, a->size * sizeof(char*));
        for (int i = (a->size)/2; i < a->size; i++) {
            a->array[i] = (char*) malloc(MAX_READ_LENGTH * sizeof(char));
        }
    }
    strcpy(a->array[a->used++], element);
}

void freeStringList(StringList *a) {
    for(int i = 0; i < a->size; i++) {
        free(a->array[i]);
    }
    free(a->array);
    a->array = NULL;
    a->used = a->size = 0;
}

int read_file(char *file_name, StringList *sequences) {
    FILE *fp;
    fp = fopen(file_name, "r");
    if(fp) {
        char *line = (char *) malloc( MAX_READ_LENGTH * sizeof(char));
        while (fgets(line, MAX_READ_LENGTH, fp) != NULL) { //A single line only
            //printf("%s", line);
            line[strcspn(line, "\n")] = 0; //Remove the trailing \n character
            insertStringList(sequences,line);
        }
        free(line);
        fclose(fp);
        return 1;
    }
    return 0; //Means error
}

//Do not use substring methods for cuda kernel, try a more primitive approach
//without memory operations for performance
//void substring(char *source, int begin_index, int end_index)
//{
//    // copy n characters from source string starting from
//    // beg index into destination
//    memmove(source, (source + begin_index), end_index-begin_index);
//    source[end_index-begin_index] = '\0';
//}

void substring(char *destination, char *source, int begin_index, int end_index)
{
    // copy n characters from source string starting from
    // beg index into destination
    memcpy(destination, (source + begin_index), end_index-begin_index);
    destination[end_index-begin_index] = '\0';
}


//* You might use these for some simple string operations in GPU
//* Put these code into your program
__device__ int d_strlen(const char* string){
    int length = 0;
    while (*string++)
        length++;
    return (length);
}

//Compares string until nth character
__device__ int d_strncmp( const char * s1, const char * s2, size_t n )
{
    while ( n && *s1 && ( *s1 == *s2 ) )
    {
        ++s1;
        ++s2;
        --n;
    }
    if ( n == 0 )
    {
        return 0;
    }
    else
    {
        return ( *(unsigned char *)s1 - *(unsigned char *)s2 );
    }
}

//__global__ void kernel_fnc(char *dev_ref, StringList *dev_str_list, int *dev_out, int k, int len_ref, int len_read){
//    char *ref_it = dev_ref;
////    printf("dev_str_list item 0: %s\n", (*dev_str_list).array[0]);
//    char *read_thread_ptr = (*dev_str_list).array[threadIdx.y] + threadIdx.x;
//
//    int l;
//    for (l = 0; l < len_ref - k + 1; l++){
//        if (d_strncmp(ref_it, read_thread_ptr, k) == 0){
//            dev_out[(len_read - k + 1) * threadIdx.y + threadIdx.x] = l;
//            break;
//        }
//
//        ref_it++;
//    }
//
//    if (l == len_ref - k + 1)
//        dev_out[(len_read - k + 1) * threadIdx.y + threadIdx.x] = -1;
//}

__global__ void kernel_fnc(char *dev_ref, char *dev_read, int *dev_out, int k, int len_ref, int len_read){
    char *ref_it = dev_ref;
//    printf("dev_str_list item 0: %s\n", (*dev_str_list).array[0]);
    char *read_thread_ptr = dev_read + threadIdx.y * len_read + threadIdx.x;

    int l;
    for (l = 0; l < len_ref - k + 1; l++){
        if (d_strncmp(ref_it, read_thread_ptr, k) == 0){
            dev_out[threadIdx.y * len_read + threadIdx.x] = l;
            break;
        }

        ref_it++;
    }

    if (l == len_ref - k + 1)
        dev_out[threadIdx.x] = -1;
}
