#include "hip/hip_runtime.h"
#include "util.h"

//void init_list(dyn_list *l, size_t initial_size){
//    l->item_arr = (int *) malloc()
//}

void initStringList(StringList *a, size_t initialSize) {
    a->array = (char**) malloc(initialSize * sizeof(char*));
    for (int i = 0; i < initialSize; i++) {
        a->array[i] = (char*) malloc(MAX_READ_LENGTH * sizeof(char));
    }
    a->used = 0;
    a->size = initialSize;
}

void insertStringList(StringList *a, char *element) {
    // a->used is the number of used entries, because a->array[a->used++] updates a->used only *after* the array has been accessed.
    // Therefore a->used can go up to a->size
    if (a->used == a->size) {
        a->size *= 2;
        a->array = (char**) realloc(a->array, a->size * sizeof(char*));
        for (int i = (a->size)/2; i < a->size; i++) {
            a->array[i] = (char*) malloc(MAX_READ_LENGTH * sizeof(char));
        }
    }
    strcpy(a->array[a->used++], element);
}

void freeStringList(StringList *a) {
    for(int i = 0; i < a->size; i++) {
        free(a->array[i]);
    }
    free(a->array);
    a->array = NULL;
    a->used = a->size = 0;
}

int read_file(char *file_name, StringList *sequences) {
    FILE *fp;
    fp = fopen(file_name, "r");
    if(fp) {
        char *line = (char *) malloc( MAX_READ_LENGTH * sizeof(char));
        while (fgets(line, MAX_READ_LENGTH, fp) != NULL) { //A single line only
            //printf("%s", line);
            line[strcspn(line, "\n")] = 0; //Remove the trailing \n character
            insertStringList(sequences,line);
        }
        free(line);
        fclose(fp);
        return 1;
    }
    return 0; //Means error
}

//Do not use substring methods for cuda kernel, try a more primitive approach
//without memory operations for performance
//void substring(char *source, int begin_index, int end_index)
//{
//    // copy n characters from source string starting from
//    // beg index into destination
//    memmove(source, (source + begin_index), end_index-begin_index);
//    source[end_index-begin_index] = '\0';
//}

void substring(char *destination, char *source, int begin_index, int end_index)
{
    // copy n characters from source string starting from
    // beg index into destination
    memcpy(destination, (source + begin_index), end_index-begin_index);
    destination[end_index-begin_index] = '\0';
}

void init_hashtable(StringList str_list, my_unordered_map *map){
    // initialize counts and lists
    for (int i = 0; i < str_list.used; ++i){
        hash_object tmp_obj;
        tmp_obj.count = 0;
        tmp_obj.index_arr = nullptr;
        tmp_obj.index_arr_index = 0;
        if ((*map).find(str_list.array[i]) == (*map).end())
            (*map)[str_list.array[i]] = tmp_obj;
    }

    // set counts
    for (int i = 0; i < str_list.used; ++i){
        (*map)[str_list.array[i]].count++;
    }

    // init index
    for (int i = 0; i < str_list.used; ++i){
        int count = (*map)[str_list.array[i]].count;
        (*map)[str_list.array[i]].index_arr = (int *) malloc(sizeof (int) * count);
//        printf("count: %d\n", count);
    }

    //set index values
    for (int i = 0; i < str_list.used; ++i){
        (*map)[str_list.array[i]].index_arr[(*map)[str_list.array[i]].index_arr_index++] = i;
    }

//    print_map(*map);
};

void print_map(my_unordered_map map){
    my_unordered_map::iterator it = map.begin();
    while (it != map.end()){
        const char *key = it->first;
        printf("[%s] : ", key);

        hash_object object = it->second;

        printf("( ");
        for (int i = 0; i < object.count; ++i) {
            printf("%d ", object.index_arr[i]);
        }

        printf(")\n");
        it++;
    }
}

void free_map(my_unordered_map map){
    my_unordered_map::iterator it = map.begin();
    while (it != map.end()){
        hash_object object = it->second;
        free(object.index_arr);
        it++;
    }
}


//* You might use these for some simple string operations in GPU
//* Put these code into your program
__device__ int d_strlen(const char* string){
    int length = 0;
    while (*string++)
        length++;
    return (length);
}

//Compares string until nth character
__device__ int d_strncmp( const char * s1, const char * s2, size_t n )
{
    while ( n && *s1 && ( *s1 == *s2 ) )
    {
        ++s1;
        ++s2;
        --n;
    }
    if ( n == 0 )
    {
        return 0;
    }
    else
    {
        return ( *(unsigned char *)s1 - *(unsigned char *)s2 );
    }
}

//__global__ void kernel_fnc(char *dev_ref, StringList *dev_str_list, int *dev_out, int k, int len_ref, int len_read){
//    printf("ENTER\n");
//    char *ref_it = dev_ref;
////    printf("dev_str_list item 0: %s\n", (*dev_str_list).array[0]);
//    char *read_thread_ptr = (*dev_str_list).array[threadIdx.y] + threadIdx.x;
//
//    int l;
//    for (l = 0; l < len_ref - k + 1; l++){
//        if (d_strncmp(ref_it, read_thread_ptr, k) == 0){
//            dev_out[(len_read - k + 1) * threadIdx.y + threadIdx.x] = l;
//            break;
//        }
//
//        ref_it++;
//    }
//
//    if (l == len_ref - k + 1)
//        dev_out[(len_read - k + 1) * threadIdx.y + threadIdx.x] = -1;
//}

__global__ void kernel_fnc(char *dev_ref, char *dev_read, int *dev_out, int k, int len_ref, int len_read){
    char *ref_it = dev_ref;
//    printf("dev_str_list item 0: %s\n", (*dev_str_list).array[0]);
    char *read_thread_ptr = dev_read + threadIdx.x;

    int l;
    for (l = 0; l < len_ref - k + 1; l++){
        if (d_strncmp(ref_it, read_thread_ptr, k) == 0){
            dev_out[threadIdx.x] = l;
            break;
        }

        ref_it++;
    }

    if (l == len_ref - k + 1)
        dev_out[threadIdx.x] = -1;
}